#include "hip/hip_runtime.h"
﻿//FOR REFERENCE ONLY

/*
*  CUDA Device Query (Runtime API) version (CUDART static linking)

Detected 1 CUDA Capable device(s)

Device 0: "NVIDIA GeForce GTX 1650"
  CUDA Driver Version / Runtime Version          12.3 / 12.0
  CUDA Capability Major/Minor version number:    7.5
  Total amount of global memory:                 4096 MBytes (4294639616 bytes)
  (14) Multiprocessors, ( 64) CUDA Cores/MP:     896 CUDA Cores
  GPU Max Clock rate:                            1515 MHz (1.51 GHz)
  Memory Clock rate:                             6001 Mhz
  Memory Bus Width:                              128-bit
  L2 Cache Size:                                 1048576 bytes
  Maximum Texture Dimension Size (x,y,z)         1D=(131072), 2D=(131072, 65536), 3D=(16384, 16384, 16384)
  Maximum Layered 1D Texture Size, (num) layers  1D=(32768), 2048 layers
  Maximum Layered 2D Texture Size, (num) layers  2D=(32768, 32768), 2048 layers
  Total amount of constant memory:               zu bytes
  Total amount of shared memory per block:       zu bytes
  Total number of registers available per block: 65536
  Warp size:                                     32
  Maximum number of threads per multiprocessor:  1024
  Maximum number of threads per block:           1024
  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
  Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
  Maximum memory pitch:                          zu bytes
  Texture alignment:                             zu bytes
  Concurrent copy and kernel execution:          Yes with 6 copy engine(s)
  Run time limit on kernels:                     Yes
  Integrated GPU sharing Host Memory:            No
  Support host page-locked memory mapping:       Yes
  Alignment requirement for Surfaces:            Yes
  Device has ECC support:                        Disabled
  CUDA Device Driver Mode (TCC or WDDM):         WDDM (Windows Display Driver Model)
  Device supports Unified Addressing (UVA):      Yes
  Device supports Compute Preemption:            Yes
  Supports Cooperative Kernel Launch:            Yes
  Supports MultiDevice Co-op Kernel Launch:      No
  Device PCI Domain ID / Bus ID / location ID:   0 / 1 / 0
  Compute Mode:
	 < Default (multiple host threads can use ::hipSetDevice() with device simultaneously) >

deviceQuery, CUDA Driver = CUDART, CUDA Driver Version = 12.3, CUDA Runtime Version = 12.0, NumDevs = 1, Device0 = NVIDIA GeForce GTX 1650
Result = PASS
*/

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define __HIPCC__
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <stdio.h>
#include <iostream>

hipError_t addWithCuda(glm::vec3* c, const glm::vec3* a, const glm::vec3* b, uint32_t size);

__global__ void addKernel(glm::vec3* c, const glm::vec3* a, const glm::vec3* b)
{
	int i = threadIdx.x;
	printf("<%.3f, %.3f, %.3f>\n", a[i].x, a[i].y, a[i].z);
	c[i] = glm::cross(a[i], b[i]);
}

int main()
{
	const uint32_t arraySize = 5;
	const glm::vec3 a[arraySize] = { {1,1,1},{2,2,2}, {3,3,3},{4,4,4}, {5,5,5} };
	const glm::vec3 b[arraySize] = { {2,5,3},{1,6,9}, {1,5,9},{5,2,1}, {8,2,6} };
	glm::vec3 c[arraySize] = { {} };

	printf("check a:\n");
	for (glm::vec3 vec : a) {
		printf("<%.3f, %.3f, %.3f>\n", vec.x, vec.y, vec.z);
	}
	printf("----\n");

	printf("check c:\n");
	for (glm::vec3 vec : c) {
		printf("<%.3f, %.3f, %.3f>\n", vec.x, vec.y, vec.z);
	}
	printf("----\n");

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	printf("-----------\n");
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for (glm::vec3 vec : c) {
		printf("<%.3f, %.3f, %.3f>\n", vec.x, vec.y, vec.z);
	}
	/*printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);*/

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	std::cin.get();

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(glm::vec3* c, const glm::vec3* a, const glm::vec3* b, uint32_t size)
{
	glm::vec3* dev_a = 0;
	glm::vec3* dev_b = 0;
	glm::vec3* dev_c = 0;
	hipError_t cudaStatus;

	printf("wrapper check: a\n");
	for (int i = 0; i < size; i++) {
		printf("<%.3f, %.3f, %.3f>\n", a[i].x, a[i].y, a[i].z);
	}
	printf("-----\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(glm::vec3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(glm::vec3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(glm::vec3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}