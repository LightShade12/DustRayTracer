#include "hip/hip_runtime.h"
#include "Core/Renderer.hpp"

#include "Core/Ray.cuh"
#include "Core/Scene/HostCamera.hpp"
#include "Core/Scene/CameraData.cuh"
#include "Core/Scene/Scene.cuh"

#include "Core/Common/CudaCommon.cuh"
#include "Kernel/RenderKernel.cuh"

#include <cuda_gl_interop.h>//for member cuda objects
#include <thrust/device_vector.h>
#include <iostream>

struct CudaAPIResource {
	CudaAPIResource() = default;
	thrust::device_vector<float3>ColorDataBuffer;
	hipGraphicsResource_t m_RenderTargetTextureCudaResource;
	hipEvent_t start, stop;
	int m_ThreadBlock_x = 8;
	int m_ThreadBlock_y = 8;
	dim3 m_BlockGridDimensions;
	dim3 m_ThreadBlockDimensions;
	SceneData m_DeviceSceneData;
	~CudaAPIResource()
	{
		ColorDataBuffer.clear();
	}
};

namespace DustRayTracer {
	PathTracerRenderer::PathTracerRenderer()
	{
		m_CudaAPIResource = new CudaAPIResource();
		m_CudaAPIResource->m_BlockGridDimensions = dim3(m_BufferWidth / m_CudaAPIResource->m_ThreadBlock_x + 1, m_BufferHeight / m_CudaAPIResource->m_ThreadBlock_y + 1);
		m_CudaAPIResource->m_ThreadBlockDimensions = dim3(m_CudaAPIResource->m_ThreadBlock_x, m_CudaAPIResource->m_ThreadBlock_y);
		hipEventCreate(&(m_CudaAPIResource->start));
		hipEventCreate(&(m_CudaAPIResource->stop));
	}

	void PathTracerRenderer::resizeResolution(uint32_t width, uint32_t height) {
		if (width == m_BufferWidth && height == m_BufferHeight)return;
		m_BufferWidth = width;
		m_BufferHeight = height;

		if (m_RenderTargetTexture_name)
		{
			m_CudaAPIResource->m_BlockGridDimensions = dim3(m_BufferWidth / m_CudaAPIResource->m_ThreadBlock_x + 1,
				m_BufferHeight / m_CudaAPIResource->m_ThreadBlock_y + 1);
			m_CudaAPIResource->m_ThreadBlockDimensions = dim3(m_CudaAPIResource->m_ThreadBlock_x, m_CudaAPIResource->m_ThreadBlock_y);

			m_CudaAPIResource->ColorDataBuffer.resize(m_BufferHeight * m_BufferWidth);

			// unregister
			hipGraphicsUnregisterResource(m_CudaAPIResource->m_RenderTargetTextureCudaResource);
			// resize
			glBindTexture(GL_TEXTURE_2D, m_RenderTargetTexture_name);
			{
				glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, m_BufferWidth, m_BufferHeight, 0, GL_RGBA, GL_FLOAT, NULL);
			}
			glBindTexture(GL_TEXTURE_2D, 0);
			// register back
			hipGraphicsGLRegisterImage(&(m_CudaAPIResource->m_RenderTargetTextureCudaResource), m_RenderTargetTexture_name, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
		}

		//Image recreation
		else
		{
			m_CudaAPIResource->m_BlockGridDimensions = dim3(m_BufferWidth / m_CudaAPIResource->m_ThreadBlock_x + 1,
				m_BufferHeight / m_CudaAPIResource->m_ThreadBlock_y + 1);
			m_CudaAPIResource->m_ThreadBlockDimensions = dim3(m_CudaAPIResource->m_ThreadBlock_x, m_CudaAPIResource->m_ThreadBlock_y);
			m_CudaAPIResource->ColorDataBuffer.resize(m_BufferHeight * m_BufferWidth);

			//GL texture configure
			glGenTextures(1, &m_RenderTargetTexture_name);
			glBindTexture(GL_TEXTURE_2D, m_RenderTargetTexture_name);

			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
			//TODO: make a switchable frame filtering mode
			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);

			glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, m_BufferWidth, m_BufferHeight, 0, GL_RGBA, GL_FLOAT, NULL);

			glBindTexture(GL_TEXTURE_2D, 0);

			hipGraphicsGLRegisterImage(&(m_CudaAPIResource->m_RenderTargetTextureCudaResource), m_RenderTargetTexture_name, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
		}

		clearAccumulation();
	}

	void PathTracerRenderer::renderFrame(float* delta)
	{
		if (m_FrameIndex == m_RendererSettings.max_samples)return;

		hipGraphicsMapResources(1, &(m_CudaAPIResource->m_RenderTargetTextureCudaResource));

		hipArray_t render_target_texture_sub_resource_array;
		hipGraphicsSubResourceGetMappedArray(&render_target_texture_sub_resource_array, (m_CudaAPIResource->m_RenderTargetTextureCudaResource), 0, 0);
		hipResourceDesc render_target_texture_resource_descriptor;
		{
			render_target_texture_resource_descriptor.resType = hipResourceTypeArray;
			render_target_texture_resource_descriptor.res.array.array = render_target_texture_sub_resource_array;
		}
		hipSurfaceObject_t render_target_texture_surface_object;
		hipCreateSurfaceObject(&render_target_texture_surface_object, &render_target_texture_resource_descriptor);

		//----
		hipEventRecord(m_CudaAPIResource->start);
		invokeRenderKernel(render_target_texture_surface_object, m_BufferWidth, m_BufferHeight,
			m_CudaAPIResource->m_BlockGridDimensions, m_CudaAPIResource->m_ThreadBlockDimensions, m_CurrentCamera.getDeviceCamera(), m_CudaAPIResource->m_DeviceSceneData, m_FrameIndex,
			thrust::raw_pointer_cast(m_CudaAPIResource->ColorDataBuffer.data()));

		checkCudaErrors(hipGetLastError());

		hipEventRecord(m_CudaAPIResource->stop);
		checkCudaErrors(hipEventSynchronize(m_CudaAPIResource->stop));

		hipEventElapsedTime(delta, m_CudaAPIResource->start, m_CudaAPIResource->stop);
		checkCudaErrors(hipDeviceSynchronize());
		//----

			//post render cuda---------------------------------------------------------------------------------
		hipDestroySurfaceObject(render_target_texture_surface_object);
		hipGraphicsUnmapResources(1, &(m_CudaAPIResource->m_RenderTargetTextureCudaResource));
		hipStreamSynchronize(0);
		m_FrameIndex++;
	}

	PathTracerRenderer::~PathTracerRenderer()
	{
		//DRT
		m_CurrentCamera.cleanup();
		//CUDA
		hipEventDestroy(m_CudaAPIResource->start);
		hipEventDestroy(m_CudaAPIResource->stop);
		delete m_CudaAPIResource;
		//GL
		glDeleteTextures(1, &m_RenderTargetTexture_name);
	}

	bool PathTracerRenderer::initialize()
	{
#ifdef DEBUG
		printf("Renderer initialized\n");
#endif // DEBUG

		return true;
	}

	bool PathTracerRenderer::shutdown()
	{
#ifdef DEBUG
		printf("Renderer shutdown sucessfully\n");
#endif // DEBUG
		return true;
	}

	void PathTracerRenderer::updateScene(DustRayTracer::HostScene& scene_object)
	{
		m_CudaAPIResource->m_DeviceSceneData.DeviceBVHNodesBuffer = thrust::raw_pointer_cast(scene_object.m_Scene->m_BVHNodesBuffer.data());
		m_CudaAPIResource->m_DeviceSceneData.DeviceBVHPrimitiveIndicesBuffer = thrust::raw_pointer_cast(scene_object.m_Scene->m_BVHTrianglesIndicesBuffer.data());
		m_CudaAPIResource->m_DeviceSceneData.DevicePrimitivesBuffer = thrust::raw_pointer_cast(scene_object.m_Scene->m_TrianglesBuffer.data());
		m_CudaAPIResource->m_DeviceSceneData.DeviceTextureBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_TexturesBuffer.data());
		m_CudaAPIResource->m_DeviceSceneData.DeviceMaterialBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_MaterialsBuffer.data());
		m_CudaAPIResource->m_DeviceSceneData.DeviceMeshBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_MeshesBuffer.data());
		m_CudaAPIResource->m_DeviceSceneData.DeviceCameraBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_CamerasBuffer.data());
		m_CudaAPIResource->m_DeviceSceneData.DeviceMeshLightsBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_TriangleLightsIndicesBuffer.data());
		//----
		m_CudaAPIResource->m_DeviceSceneData.DeviceMeshBufferSize = scene_object.m_Scene->m_MeshesBuffer.size();
		m_CudaAPIResource->m_DeviceSceneData.DevicePrimitivesBufferSize = scene_object.m_Scene->m_TrianglesBuffer.size();
		m_CudaAPIResource->m_DeviceSceneData.DeviceBVHPrimitiveIndicesBufferSize = scene_object.m_Scene->m_BVHTrianglesIndicesBuffer.size();
		m_CudaAPIResource->m_DeviceSceneData.DeviceMeshLightsBufferSize = scene_object.m_Scene->m_TriangleLightsIndicesBuffer.size();
		m_CudaAPIResource->m_DeviceSceneData.DeviceBVHNodesBufferSize = scene_object.m_Scene->m_BVHNodesBuffer.size();
		m_CudaAPIResource->m_DeviceSceneData.DeviceCameraBufferSize = scene_object.m_Scene->m_CamerasBuffer.size();
		//---
		DustRayTracer::CameraData* cam = &scene_object.m_Scene->m_CamerasBuffer[0];
		m_CurrentCamera = HostCamera(thrust::raw_pointer_cast(cam));
		m_CudaAPIResource->m_DeviceSceneData.RenderSettings = m_RendererSettings;
	}

	void PathTracerRenderer::updateRendererConfig(const RendererSettings& config)
	{
		{ m_RendererSettings = config; m_CudaAPIResource->m_DeviceSceneData.RenderSettings = m_RendererSettings; }
	}

	/*HostCamera PathTracerRenderer::getCamera() const
	{
		return m_CurrentCamera;
	}*/

	DustRayTracer::HostCamera* PathTracerRenderer::getCameraPtr()
	{
		return &m_CurrentCamera;
	}

	void PathTracerRenderer::changeCamera(uint32_t camera_idx)
	{
		if (camera_idx >= m_CudaAPIResource->m_DeviceSceneData.DeviceCameraBufferSize)camera_idx = m_CudaAPIResource->m_DeviceSceneData.DeviceCameraBufferSize - 1;
		else if (camera_idx < 0)camera_idx = 0;
		DustRayTracer::CameraData* cm = &m_CudaAPIResource->m_DeviceSceneData.DeviceCameraBufferPtr[camera_idx];
		m_CurrentCamera = HostCamera(thrust::raw_pointer_cast(cm));
	}

	void PathTracerRenderer::clearAccumulation()
	{
		thrust::fill(m_CudaAPIResource->ColorDataBuffer.begin(), m_CudaAPIResource->ColorDataBuffer.end(), make_float3(0, 0, 0));
		m_FrameIndex = 1;
	}
}