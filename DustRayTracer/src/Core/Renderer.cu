#include "hip/hip_runtime.h"
#include "Core/Renderer.hpp"

#include "Core/Ray.cuh"
#include "Core/Scene/HostCamera.hpp"
#include "Core/Scene/CameraData.cuh"
#include "Core/Scene/Scene.cuh"

#include "Core/Common/CudaCommon.cuh"
#include "Kernel/RenderKernel.cuh"

#include <cuda_gl_interop.h>//for member cuda objects
#include <thrust/device_vector.h>
#include <iostream>

struct ThrustRGB32FBufferWrapper
{
	thrust::device_vector<float3>ColorDataBuffer;

	~ThrustRGB32FBufferWrapper()
	{
		ColorDataBuffer.clear();
	}
};

struct CudaGLAPI {
	CudaGLAPI() = default;
	hipGraphicsResource_t m_RenderTargetTextureCudaResource;
	hipEvent_t start, stop;
};

namespace DustRayTracer {
	PathTracerRenderer::PathTracerRenderer()
	{
		m_AccumulationFrameBuffer = new ThrustRGB32FBufferWrapper();
		m_CudaGLAPI = new CudaGLAPI();
		m_BlockGridDimensions = dim3(m_BufferWidth / m_ThreadBlock_x + 1, m_BufferHeight / m_ThreadBlock_y + 1);
		m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);
		hipEventCreate(&(m_CudaGLAPI->start));
		hipEventCreate(&(m_CudaGLAPI->stop));
	}

	void PathTracerRenderer::resizeResolution(uint32_t width, uint32_t height) {
		if (width == m_BufferWidth && height == m_BufferHeight)return;
		m_BufferWidth = width;
		m_BufferHeight = height;

		if (m_RenderTargetTexture_name)
		{
			m_BlockGridDimensions = dim3(m_BufferWidth / m_ThreadBlock_x + 1, m_BufferHeight / m_ThreadBlock_y + 1);
			m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);

			m_AccumulationFrameBuffer->ColorDataBuffer.resize(m_BufferHeight * m_BufferWidth);

			// unregister
			hipGraphicsUnregisterResource(m_CudaGLAPI->m_RenderTargetTextureCudaResource);
			// resize
			glBindTexture(GL_TEXTURE_2D, m_RenderTargetTexture_name);
			{
				glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, m_BufferWidth, m_BufferHeight, 0, GL_RGBA, GL_FLOAT, NULL);
			}
			glBindTexture(GL_TEXTURE_2D, 0);
			// register back
			hipGraphicsGLRegisterImage(&(m_CudaGLAPI->m_RenderTargetTextureCudaResource), m_RenderTargetTexture_name, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
		}

		//Image recreation
		else
		{
			m_BlockGridDimensions = dim3(m_BufferWidth / m_ThreadBlock_x + 1, m_BufferHeight / m_ThreadBlock_y + 1);
			m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);
			m_AccumulationFrameBuffer->ColorDataBuffer.resize(m_BufferHeight * m_BufferWidth);

			//GL texture configure
			glGenTextures(1, &m_RenderTargetTexture_name);
			glBindTexture(GL_TEXTURE_2D, m_RenderTargetTexture_name);

			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
			//TODO: make a switchable frame filtering mode
			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
			glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);

			glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, m_BufferWidth, m_BufferHeight, 0, GL_RGBA, GL_FLOAT, NULL);

			glBindTexture(GL_TEXTURE_2D, 0);

			hipGraphicsGLRegisterImage(&(m_CudaGLAPI->m_RenderTargetTextureCudaResource), m_RenderTargetTexture_name, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
		}

		clearAccumulation();
	}

	void PathTracerRenderer::renderFrame(float* delta)
	{
		if (m_FrameIndex == m_RendererSettings.max_samples)return;

		hipGraphicsMapResources(1, &(m_CudaGLAPI->m_RenderTargetTextureCudaResource));

		hipArray_t render_target_texture_sub_resource_array;
		hipGraphicsSubResourceGetMappedArray(&render_target_texture_sub_resource_array, (m_CudaGLAPI->m_RenderTargetTextureCudaResource), 0, 0);
		hipResourceDesc render_target_texture_resource_descriptor;
		{
			render_target_texture_resource_descriptor.resType = hipResourceTypeArray;
			render_target_texture_resource_descriptor.res.array.array = render_target_texture_sub_resource_array;
		}
		hipSurfaceObject_t render_target_texture_surface_object;
		hipCreateSurfaceObject(&render_target_texture_surface_object, &render_target_texture_resource_descriptor);

		//----
		hipEventRecord(m_CudaGLAPI->start);
		invokeRenderKernel(render_target_texture_surface_object, m_BufferWidth, m_BufferHeight,
			m_BlockGridDimensions, m_ThreadBlockDimensions, m_CurrentCamera.getDeviceCamera(), m_DeviceSceneData, m_FrameIndex,
			thrust::raw_pointer_cast(m_AccumulationFrameBuffer->ColorDataBuffer.data()));

		checkCudaErrors(hipGetLastError());

		hipEventRecord(m_CudaGLAPI->stop);
		checkCudaErrors(hipEventSynchronize(m_CudaGLAPI->stop));

		hipEventElapsedTime(delta, m_CudaGLAPI->start, m_CudaGLAPI->stop);
		checkCudaErrors(hipDeviceSynchronize());
		//----

			//post render cuda---------------------------------------------------------------------------------
		hipDestroySurfaceObject(render_target_texture_surface_object);
		hipGraphicsUnmapResources(1, &(m_CudaGLAPI->m_RenderTargetTextureCudaResource));
		hipStreamSynchronize(0);
		m_FrameIndex++;
	}

	PathTracerRenderer::~PathTracerRenderer()
	{
		//DRT
		m_CurrentCamera.cleanup();
		//CUDA
		delete m_AccumulationFrameBuffer;
		hipEventDestroy(m_CudaGLAPI->start);
		hipEventDestroy(m_CudaGLAPI->stop);
		delete m_CudaGLAPI;
		//GL
		glDeleteTextures(1, &m_RenderTargetTexture_name);
	}

	bool PathTracerRenderer::initialize()
	{
#ifdef DEBUG
		printf("Renderer initialized\n");
#endif // DEBUG

		return true;
	}

	bool PathTracerRenderer::shutdown()
	{
#ifdef DEBUG
		printf("Renderer shutdown sucessfully\n");
#endif // DEBUG
		return true;
	}

	void PathTracerRenderer::updateScene(DustRayTracer::HostScene& scene_object)
	{
		m_DeviceSceneData.DeviceBVHNodesBuffer = thrust::raw_pointer_cast(scene_object.m_Scene->m_BVHNodesBuffer.data());
		m_DeviceSceneData.DeviceBVHPrimitiveIndicesBuffer = thrust::raw_pointer_cast(scene_object.m_Scene->m_BVHTrianglesIndicesBuffer.data());
		m_DeviceSceneData.DevicePrimitivesBuffer = thrust::raw_pointer_cast(scene_object.m_Scene->m_TrianglesBuffer.data());
		m_DeviceSceneData.DeviceTextureBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_TexturesBuffer.data());
		m_DeviceSceneData.DeviceMaterialBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_MaterialsBuffer.data());
		m_DeviceSceneData.DeviceMeshBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_MeshesBuffer.data());
		m_DeviceSceneData.DeviceCameraBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_CamerasBuffer.data());
		m_DeviceSceneData.DeviceMeshLightsBufferPtr = thrust::raw_pointer_cast(scene_object.m_Scene->m_TriangleLightsIndicesBuffer.data());
		//----
		m_DeviceSceneData.DeviceMeshBufferSize = scene_object.m_Scene->m_MeshesBuffer.size();
		m_DeviceSceneData.DevicePrimitivesBufferSize = scene_object.m_Scene->m_TrianglesBuffer.size();
		m_DeviceSceneData.DeviceBVHPrimitiveIndicesBufferSize = scene_object.m_Scene->m_BVHTrianglesIndicesBuffer.size();
		m_DeviceSceneData.DeviceMeshLightsBufferSize = scene_object.m_Scene->m_TriangleLightsIndicesBuffer.size();
		m_DeviceSceneData.DeviceBVHNodesBufferSize = scene_object.m_Scene->m_BVHNodesBuffer.size();
		m_DeviceSceneData.DeviceCameraBufferSize = scene_object.m_Scene->m_CamerasBuffer.size();
		//---
		DustRayTracer::CameraData* cam = &scene_object.m_Scene->m_CamerasBuffer[0];
		m_CurrentCamera = HostCamera(thrust::raw_pointer_cast(cam));
		m_DeviceSceneData.RenderSettings = m_RendererSettings;
	}

	/*HostCamera PathTracerRenderer::getCamera() const
	{
		return m_CurrentCamera;
	}*/

	DustRayTracer::HostCamera* PathTracerRenderer::getCameraPtr()
	{
		return &m_CurrentCamera;
	}

	void PathTracerRenderer::changeCamera(uint32_t camera_idx)
	{
		if (camera_idx >= m_DeviceSceneData.DeviceCameraBufferSize)camera_idx = m_DeviceSceneData.DeviceCameraBufferSize - 1;
		else if (camera_idx < 0)camera_idx = 0;
		DustRayTracer::CameraData* cm = &m_DeviceSceneData.DeviceCameraBufferPtr[camera_idx];
		m_CurrentCamera = HostCamera(thrust::raw_pointer_cast(cm));
	}

	void PathTracerRenderer::clearAccumulation()
	{
		thrust::fill(m_AccumulationFrameBuffer->ColorDataBuffer.begin(), m_AccumulationFrameBuffer->ColorDataBuffer.end(), make_float3(0, 0, 0));
		m_FrameIndex = 1;
	}
}