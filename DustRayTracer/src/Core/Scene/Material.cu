#include "Material.cuh"
#include <hip/hip_runtime.h>

namespace DustRayTracer {
	HostMaterial::HostMaterial(MaterialData* device_material_data)
	{
		if (device_material_data) {
			m_DeviceMaterialData = device_material_data;
			hipMemcpy(&m_HostMaterialData, m_DeviceMaterialData, sizeof(MaterialData), hipMemcpyDeviceToHost);
		}
	}

	void HostMaterial::updateDevice() {
		if (m_DeviceMaterialData) {
			hipMemcpy(m_DeviceMaterialData, &m_HostMaterialData, sizeof(MaterialData), hipMemcpyHostToDevice);
		}
	}
}