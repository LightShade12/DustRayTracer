#include "hip/hip_runtime.h"
#include "RenderKernel.cuh"

#include "Core/Scene/Scene.cuh"
#include "Core/Scene/SceneData.cuh"
#include "Core/PostProcess.cuh"
#include "Shaders/RayGen.cuh"

#include <hip/hip_runtime.h>
#include <>
#define __HIPCC__ // used to get surf2d indirect functions;not how it should be done
#include <surface_indirect_functions.h>

__global__ void integratorKernel(hipSurfaceObject_t surface_object, int max_x, int max_y,
	const DustRayTracer::CameraData* device_camera, uint32_t frameidx, float3* accumulation_buffer, const SceneData scenedata);

//TODO: fix inconsistent buffer and primitive-triangle naming
void invokeRenderKernel(
	hipSurfaceObject_t surfaceobj, uint32_t width, uint32_t height,
	dim3 _blocks, dim3 _threads, const DustRayTracer::CameraData* device_camera,
	const SceneData& scene_data, uint32_t frameidx, float3* accumulation_buffer)
{
	integratorKernel << < _blocks, _threads >> > (surfaceobj, width, height, device_camera, frameidx, accumulation_buffer, scene_data);
}

//Monte Carlo Render Kernel
__global__ void integratorKernel(hipSurfaceObject_t surface_object, int max_x, int max_y, const DustRayTracer::CameraData* device_camera, 
	uint32_t frameidx, float3* accumulation_buffer, const SceneData scenedata)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= max_x) || (j >= max_y)) return;

	//raygen is the integration solver, the renderloop is integrator
	float3 sampled_radiance = rayGen(i, j, max_x, max_y, device_camera, frameidx, scenedata);//just some encapsulation; generally raygen is integrator

	//Monte carlo
	accumulation_buffer[i + j * max_x] += sampled_radiance;
	float3 estimated_radiance = accumulation_buffer[i + j * max_x] / frameidx;

	float3 processed_radiance = estimated_radiance;

	if (scenedata.RenderSettings.RenderMode == RendererSettings::RenderModes::NORMALMODE || scenedata.RenderSettings.DebugMode == RendererSettings::DebugModes::ALBEDO_DEBUG)
	{
		//order matters
		if (scenedata.RenderSettings.enable_tone_mapping)processed_radiance = toneMapping(processed_radiance, device_camera->exposure);
		if (scenedata.RenderSettings.enable_gamma_correction)processed_radiance = gammaCorrection(processed_radiance);//inverse EOTF
	}

	float4 color = { processed_radiance.x, processed_radiance.y, processed_radiance.z, 1 };

	surf2Dwrite(color, surface_object, i * (int)sizeof(float4), j);//has to be uchar4/2/1 or float4/2/1; no 3 comp color
};


