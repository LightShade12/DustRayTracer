#include "TraceRay.cuh"

#include "Core/Scene/Scene.cuh"
#include "Core/BVH/BVHTraversal.cuh"

#include "Shaders/ClosestHit.cuh"
#include "Shaders/Miss.cuh"
#include "Shaders/Intersection.cuh"
#include "Shaders/AnyHit.cuh"
#include "Shaders/Debug.cuh"

#include <hip/hip_vector_types.h>

//traverse accel struct
__device__ HitPayload TraceRay(const Ray& ray, const SceneData* scenedata) {
	float closestHitDistance = FLT_MAX;//closesthit
	HitPayload workingPayload;
	const Triangle* hitprim = nullptr;
	bool debug = false;

	//here working payload is being sent in as closest hit payload
	workingPayload.hit_distance = FLT_MAX;
	traverseBVH(ray, (scenedata->DeviceBVHNodesBufferSize) - 1, &workingPayload, debug, scenedata);
	closestHitDistance = workingPayload.hit_distance;
	hitprim = workingPayload.primitiveptr;

	if (debug)
		return Debug();

	//Have not hit
	if (hitprim == nullptr)
	{
		return Miss(ray, workingPayload.color);
	}

	return ClosestHit(ray, closestHitDistance, hitprim, workingPayload.color);
}

//does not support glass material; cuz no mat processing
__device__ bool RayTest(const Ray& ray, const SceneData* scenedata)
{
	return traverseBVH_raytest(ray, (scenedata->DeviceBVHNodesBufferSize) - 1, scenedata);
}