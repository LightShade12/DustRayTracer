#include "hip/hip_runtime.h"
#include "ImportanceSampler.cuh"

#include "BRDF.cuh"
#include "Scene/Material.cuh"
#include "Common/physical_units.hpp"

#include "Core/CudaMath/Random.cuh"
#include "Core/CudaMath/helper_math.cuh"

__device__ float3 sampleGGX(float3 normal, float roughness, float2 xi) {
	float alpha = roughness * roughness;

	float phi = 2.0f * PI * xi.x;
	float cosTheta = sqrtf((1.0f - xi.y) / (1.0f + (alpha * alpha - 1.0f) * xi.y));
	float sinTheta = sqrtf(1.0f - cosTheta * cosTheta);

	float3 H;
	H.x = sinTheta * cosf(phi);
	H.y = sinTheta * sinf(phi);
	H.z = cosTheta;

	float3 up = fabs(normal.z) < 0.999 ? make_float3(0.0, 0.0, 1.0) : make_float3(1.0, 0.0, 0.0);
	float3 tangent = normalize(cross(up, normal));
	float3 bitangent = cross(normal, tangent);

	return normalize(tangent * H.x + bitangent * H.y + normal * H.z);
}
__device__ float3 sampleCosineWeightedHemisphere(float3 normal, float2 xi) {
	// Generate a cosine-weighted direction in the local frame
	float phi = 2.0f * PI * xi.x;
	float cosTheta = sqrtf(xi.y);//TODO: might have to switch with sinTheta
	float sinTheta = sqrtf(1.0f - xi.y);

	float3 H;
	H.x = sinTheta * cosf(phi);
	H.y = sinTheta * sinf(phi);
	H.z = cosTheta;

	// Create an orthonormal basis (tangent, bitangent, normal)
	float3 up = fabs(normal.z) < 0.999f ? make_float3(0.0f, 0.0f, 1.0f) : make_float3(1.0f, 0.0f, 0.0f);
	float3 tangent = normalize(cross(up, normal));
	float3 bitangent = cross(normal, tangent);

	// Transform the sample direction from local space to world space
	return normalize(tangent * H.x + bitangent * H.y + normal * H.z);
}

__device__ float3 importanceSampleBRDF(float3 normal, float3 viewDir, const Material& material, uint32_t& seed, float& pdf, const SceneData& scene_data, float2 texture_uv) {
	float roughness = material.Roughness;
	float metallicity = material.Metallicity;
	float3 H{};
	float3 sampleDir;

	float random_value = randomFloat(seed);
	float2 xi = make_float2(randomFloat(seed), randomFloat(seed));//uniform rng sample

	//if (random_value < metallicity)
	if (false)
	{
		// Metallic (Specular only)
		H = sampleGGX(normal, roughness, xi);
		sampleDir = reflect(-viewDir, H);
		pdf = D_GGX(dot(normal, H), roughness) * dot(normal, H) / (4.0f * dot(sampleDir, H));
	}
	else {
		// Non-metallic

		//diffuse
		sampleDir = sampleCosineWeightedHemisphere(normal, xi);
		pdf = dot(normal, sampleDir) * (1.0f / PI);
	}

	return normalize(sampleDir);
}