#include "hip/hip_runtime.h"
#include "BRDF.cuh"

#include "Scene/Texture.cuh"
#include "Scene/Triangle.cuh"
#include "Scene/Mesh.cuh"
#include "BVH/BVHNode.cuh"
#include "Scene/Material.cuh"
#include "Scene/SceneData.cuh"

#include "Core/CudaMath/helper_math.cuh"
#include "Common/physical_units.hpp"
#include <thrust/device_vector.h>

__device__ float fresnelSchlick90(float cosTheta, float F0, float F90) {
	return F0 + (F90 - F0) * pow(1.0 - cosTheta, 5.0);
}

__device__ float disneyDiffuseFactor(float NoV, float NoL, float VoH, float roughness) {
	float alpha = roughness * roughness;
	float F90 = 0.5 + 2.0 * alpha * VoH * VoH;
	float F_in = fresnelSchlick90(NoL, 1.0, F90);
	float F_out = fresnelSchlick90(NoV, 1.0, F90);
	return F_in * F_out;
}

__device__ float3 fresnelSchlick(float VoH, float3 F0) {
	float3 F = F0 + (1.0 - F0) * pow(1.0 - VoH, 5.0);
	return clamp(F, 0, 1);
}

__device__ float D_GGX(float NoH, float roughness) {
	float clamp_rough = fmaxf(roughness, 0.02f);
	float alpha = clamp_rough * clamp_rough;
	float alpha2 = alpha * alpha;
	float NoH2 = NoH * NoH;
	float b = (NoH2 * (alpha2 - 1.0) + 1.0);//alt: NoH2 * alpha2 + (1 - NoH2)
	//float b = NoH2 * alpha2 + (1 - NoH2);//alt: NoH2 * alpha2 + (1 - NoH2)
	return alpha2 / (PI * (b * b));
}

//maybe wrong
__device__ float G1_GGX_Schlick(float NoV, float roughness) {
	float alpha = roughness * roughness;
	float k = alpha / 2.0;
	return NoV / (NoV * (1.0 - k) + k);
}

__device__ float G_Smith(float NoV, float NoL, float roughness) {
	return G1_GGX_Schlick(NoL, roughness) * G1_GGX_Schlick(NoV, roughness);
}

//combined diffuse+specular brdf
__device__ float3 BRDF(float3 incoming_lightdir, float3 outgoing_viewdir, float3 normal, const SceneData& scene_data,
	const Material& material, const float2& texture_uv)
{
	float3 H = normalize(outgoing_viewdir + incoming_lightdir);

	float NoV = clamp(dot(normal, outgoing_viewdir), 0.0, 1.0);//TODO:change to maxf
	float NoL = clamp(dot(normal, incoming_lightdir), 0.0, 1.0);
	float NoH = clamp(dot(normal, H), 0.0, 1.0);
	float VoH = clamp(dot(outgoing_viewdir, H), 0.0, 1.0);

	float reflectance = material.Reflectance;
	float roughness = material.Roughness;
	float metallicity = material.Metallicity;
	float3 baseColor = material.Albedo;

	if (material.AlbedoTextureIndex >= 0)baseColor = scene_data.DeviceTextureBufferPtr[material.AlbedoTextureIndex].getPixel(texture_uv);
	//roughness-metallic texture
	if (material.RoughnessTextureIndex >= 0) {
		float3 col = scene_data.DeviceTextureBufferPtr[material.RoughnessTextureIndex].getPixel(texture_uv, true);
		roughness = col.y;
		metallicity = col.z;
	}

	if (scene_data.RenderSettings.UseMaterialOverride)
	{
		reflectance = scene_data.RenderSettings.OverrideMaterial.Reflectance;
		roughness = scene_data.RenderSettings.OverrideMaterial.Roughness;
		metallicity = scene_data.RenderSettings.OverrideMaterial.Metallicity;
		baseColor = scene_data.RenderSettings.OverrideMaterial.Albedo;
	}

	float3 f0 = make_float3(0.16 * (reflectance * reflectance));//f0=0.04 for most mats
	f0 = lerp(f0, baseColor, metallicity);
	float3 F = fresnelSchlick(VoH, f0);

	float3 spec = make_float3(0);

	if (NoL > 0 && VoH > 0) {
		float G = G_Smith(NoV, NoL, roughness);
		float D = D_GGX(NoH, roughness);
		spec = (F * D * G) / (4.0 * fmaxf(NoV, 0.0001) * NoL);//maybe clamp NOV?
	}

	float3 rhoD = baseColor;

	rhoD *= (1.0 - F);//F=Ks
	//rhoD *= disneyDiffuseFactor(NoV, NoL, VoH, roughness);	// optionally for less AO
	rhoD *= (1.0 - metallicity);

	float3 diff = rhoD / PI;
	diff *= NoL;//NoL is lambert falloff
	return diff + spec;
	//return diff;
}

__device__ float ImportanceSampleGGX_VNDF_PDF(float roughness, float3 N, float3 V, float3 L)
{
	float clamped_roughness = fmaxf(roughness, 0.01);
	float3 H = normalize(L + V);
	float NoH = clamp(dot(N, H), 0.f, 1.f);
	float VoH = clamp(dot(V, H), 0.f, 1.f);
	float alpha = clamped_roughness * clamped_roughness;
	float alpha2 = alpha * alpha;
	float NoH2 = NoH * NoH;

	//float b = (NoH2 * (alpha2 - 1.0) + 1.0);
	float b = NoH2 * alpha2 + (1 - NoH2);

	float D = alpha2 / (PI * (b * b));

	return (VoH > 0.0) ? D / (4.0 * VoH) : 0.0;
}