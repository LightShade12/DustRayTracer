#include "core/Renderer/Renderer.hpp"
#include "core/Renderer/private/Camera/Camera.cuh"

#include "Kernel/RenderKernel.cuh"

#include <iostream>

//prints error code
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

Renderer::Renderer()
{
	blocks = dim3(m_BufferWidth / tx + 1, m_BufferHeight / ty + 1);
	threads = dim3(tx, ty);
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

void Renderer::ResizeBuffer(uint32_t width, uint32_t height) {

	if (width == m_BufferWidth && height == m_BufferHeight)return;
	m_BufferWidth = width;
	m_BufferHeight = height;

	if (m_RenderTarget_name)
	{
		blocks = dim3(m_BufferWidth / tx + 1, m_BufferHeight / ty + 1);
		threads = dim3(tx, ty);

		// unregister
		hipGraphicsUnregisterResource(m_viewCudaResource);
		// resize
		glBindTexture(GL_TEXTURE_2D, m_RenderTarget_name);
		{
			glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, m_BufferWidth, m_BufferHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
		}
		glBindTexture(GL_TEXTURE_2D, 0);
		// register back
		hipGraphicsGLRegisterImage(&m_viewCudaResource, m_RenderTarget_name, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
	}

	//Image recreation
	else
	{
		blocks = dim3(m_BufferWidth / tx + 1, m_BufferHeight / ty + 1);
		threads = dim3(tx, ty);
		//GL texture configure
		glGenTextures(1, &m_RenderTarget_name);
		glBindTexture(GL_TEXTURE_2D, m_RenderTarget_name);

		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, m_BufferWidth, m_BufferHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

		glBindTexture(GL_TEXTURE_2D, 0);

		hipGraphicsGLRegisterImage(&m_viewCudaResource, m_RenderTarget_name, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);

	}
}

void Renderer::Render(Camera* cam, const Scene& scene, float* delta)
{

	hipGraphicsMapResources(1, &m_viewCudaResource);

	hipArray_t viewCudaArray;
	hipGraphicsSubResourceGetMappedArray(&viewCudaArray, m_viewCudaResource, 0, 0);
	hipResourceDesc viewCudaArrayResourceDesc;
	{
		viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
		viewCudaArrayResourceDesc.res.array.array = viewCudaArray;
	}
	hipSurfaceObject_t viewCudaSurfaceObject;
	hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc);
	
	//----
	hipEventRecord(start);

	InvokeRenderKernel(viewCudaSurfaceObject, m_BufferWidth, m_BufferHeight, blocks, threads,cam,scene,m_FrameIndex);
	checkCudaErrors(hipGetLastError());
	
	hipEventRecord(stop);
	checkCudaErrors(hipEventSynchronize(stop));
	
	hipEventElapsedTime(delta, start, stop);
	//checkCudaErrors(hipDeviceSynchronize());
	//----

	//post render cuda---------------------------------------------------------------------------------
	hipDestroySurfaceObject(viewCudaSurfaceObject);
	hipGraphicsUnmapResources(1, &m_viewCudaResource);
	hipStreamSynchronize(0);
	m_FrameIndex++;
}

GLuint& Renderer::GetRenderTargetImage_name()
{
	return m_RenderTarget_name;
}

Renderer::~Renderer()
{
	hipEventDestroy(start);
	hipEventDestroy(stop);
	glDeleteTextures(1, &m_RenderTarget_name);
}
