#include "hip/hip_runtime.h"
#include "BVHBuilder.cuh"

BVHNode* BVHBuilder::build(const thrust::universal_vector<Triangle>& primitives)
{
	std::shared_ptr<BVHNode>hostBVHroot = std::make_shared<BVHNode>();

	printf("root prim count:%zu \n", primitives.size());

	float3 minextent;
	float3 extent = getAbsoluteExtent(thrust::raw_pointer_cast(primitives.data()),
		primitives.size(), minextent);//error prone
	hostBVHroot->m_BoundingBox = Bounds3f(minextent, minextent + extent);

	hostBVHroot->primitives_count = primitives.size();

	//if leaf candidate
	if (primitives.size() <= m_TargetLeafPrimitivesCount)
	{
		hostBVHroot->m_IsLeaf = true;
		std::vector<const Triangle*>DevToHostPrimitivePtrs;
		for (size_t i = 0; i < primitives.size(); i++)
		{
			DevToHostPrimitivePtrs.push_back(&(primitives[i]));
		}
		hipMallocManaged(&(hostBVHroot->dev_primitive_ptrs_buffer), sizeof(const Triangle*) * DevToHostPrimitivePtrs.size());
		hipMemcpy(hostBVHroot->dev_primitive_ptrs_buffer, DevToHostPrimitivePtrs.data(), sizeof(const Triangle*) * DevToHostPrimitivePtrs.size(), hipMemcpyHostToDevice);

		BVHNode* deviceBVHroot;
		hipMallocManaged(&deviceBVHroot, sizeof(BVHNode));
		hipMemcpy(deviceBVHroot, hostBVHroot.get(), sizeof(BVHNode), hipMemcpyHostToDevice);

		printf("made root leaf with %d prims\n", hostBVHroot->primitives_count);
		return deviceBVHroot;
	}

	std::shared_ptr<BVHNode>left = std::make_shared<BVHNode>();
	std::shared_ptr<BVHNode>right = std::make_shared<BVHNode>();

	//TODO: candidate for for_each loop
	thrust::host_vector<const Triangle*>dev_prim_ptrs;
	dev_prim_ptrs.reserve(primitives.size());
	for (size_t i = 0; i < primitives.size(); i++)
	{
		dev_prim_ptrs.push_back(&(primitives[i]));
	}

	makePartition(dev_prim_ptrs.data(),
		primitives.size(), *(left), *(right));

	recursiveBuild(*left);
	recursiveBuild(*right);

	hipMallocManaged(&hostBVHroot->dev_child1, sizeof(BVHNode));
	hipMemcpy(hostBVHroot->dev_child1, left.get(), sizeof(BVHNode), hipMemcpyHostToDevice);

	hipMallocManaged(&hostBVHroot->dev_child2, sizeof(BVHNode));
	hipMemcpy(hostBVHroot->dev_child2, right.get(), sizeof(BVHNode), hipMemcpyHostToDevice);

	BVHNode* deviceBVHroot;
	hipMallocManaged(&deviceBVHroot, sizeof(BVHNode));
	hipMemcpy(deviceBVHroot, hostBVHroot.get(), sizeof(BVHNode), hipMemcpyHostToDevice);

	return deviceBVHroot;
}

void BVHBuilder::recursiveBuild(BVHNode& node)
{
	printf("recursive build, child node prim count: %d \n", node.primitives_count);
	if (node.primitives_count <= m_TargetLeafPrimitivesCount)
	{
		printf("made a leaf node with %d prims---------------\n", node.primitives_count);
		node.m_IsLeaf = true; return;
	}
	else
	{
		std::shared_ptr<BVHNode>leftnode = std::make_shared<BVHNode>();
		std::shared_ptr<BVHNode>rightnode = std::make_shared<BVHNode>();

		makePartition(node.dev_primitive_ptrs_buffer, node.primitives_count, *leftnode, *rightnode);
		hipFree(node.dev_primitive_ptrs_buffer);
		//checkCudaErrors(hipGetLastError());
		node.dev_primitive_ptrs_buffer = nullptr;

		recursiveBuild(*leftnode);
		recursiveBuild(*rightnode);

		hipMallocManaged(&node.dev_child1, sizeof(BVHNode));
		hipMemcpy(node.dev_child1, leftnode.get(), sizeof(BVHNode), hipMemcpyHostToDevice);

		hipMallocManaged(&node.dev_child2, sizeof(BVHNode));
		hipMemcpy(node.dev_child2, rightnode.get(), sizeof(BVHNode), hipMemcpyHostToDevice);
	}
}

void BVHBuilder::binToNodes(BVHNode& left, BVHNode& right, float bin, PARTITION_AXIS axis, const Triangle** primitives_ptrs_buffer, size_t primitives_count)
{
	std::vector<const Triangle*>left_prim_ptrs;
	std::vector<const Triangle*>right_prim_ptrs;

	//sorting
	//TODO: candidate for foreach
	for (size_t primidx = 0; primidx < primitives_count; primidx++)
	{
		const Triangle* triangle = (primitives_ptrs_buffer[primidx]);

		switch (axis)
		{
		case BVHBuilder::PARTITION_AXIS::X_AXIS:
			if (triangle->centroid.x < bin)left_prim_ptrs.push_back(triangle);
			else right_prim_ptrs.push_back(triangle);
			break;
		case BVHBuilder::PARTITION_AXIS::Y_AXIS:
			if (triangle->centroid.y < bin)left_prim_ptrs.push_back(triangle);
			else right_prim_ptrs.push_back(triangle);
			break;
		case BVHBuilder::PARTITION_AXIS::Z_AXIS:
			if (triangle->centroid.z < bin)left_prim_ptrs.push_back(triangle);
			else right_prim_ptrs.push_back(triangle);
			break;
		default:
			break;
		}
	}

	left.primitives_count = left_prim_ptrs.size();
	size_t buffersize = sizeof(const Triangle*) * left_prim_ptrs.size();
	hipMallocManaged(&(left.dev_primitive_ptrs_buffer), buffersize);
	hipMemcpy(left.dev_primitive_ptrs_buffer, left_prim_ptrs.data(), buffersize, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());
	float3 leftminextent;
	float3 leftextent = getAbsoluteExtent(left_prim_ptrs.data(), left.primitives_count, leftminextent);//diff args good for error checking
	left.m_BoundingBox = Bounds3f(leftminextent, leftminextent + leftextent);

	right.primitives_count = right_prim_ptrs.size();
	buffersize = sizeof(const Triangle*) * right_prim_ptrs.size();
	hipMallocManaged(&(right.dev_primitive_ptrs_buffer), buffersize);
	hipMemcpy(right.dev_primitive_ptrs_buffer, right_prim_ptrs.data(), buffersize, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());
	float3 rightminextent;
	float3 rightextent = getAbsoluteExtent(right.dev_primitive_ptrs_buffer, right.primitives_count, rightminextent);
	right.m_BoundingBox = Bounds3f(rightminextent, rightminextent + rightextent);
}

void BVHBuilder::binToShallowNodes(BVHNode& left, BVHNode& right, float bin, PARTITION_AXIS axis, const Triangle** primitives_ptrs_buffer, size_t primitives_count)
{
	std::vector<const Triangle*>left_prim_ptrs;
	std::vector<const Triangle*>right_prim_ptrs;

	//sorting
	//TODO: candidate for foreach
	for (size_t primidx = 0; primidx < primitives_count; primidx++)
	{
		const Triangle* triangle = (primitives_ptrs_buffer[primidx]);

		switch (axis)
		{
		case BVHBuilder::PARTITION_AXIS::X_AXIS:
			if (triangle->centroid.x < bin)left_prim_ptrs.push_back(triangle);
			else right_prim_ptrs.push_back(triangle);
			break;
		case BVHBuilder::PARTITION_AXIS::Y_AXIS:
			if (triangle->centroid.y < bin)left_prim_ptrs.push_back(triangle);
			else right_prim_ptrs.push_back(triangle);
			break;
		case BVHBuilder::PARTITION_AXIS::Z_AXIS:
			if (triangle->centroid.z < bin)left_prim_ptrs.push_back(triangle);
			else right_prim_ptrs.push_back(triangle);
			break;
		default:
			break;
		}
	}

	left.primitives_count = left_prim_ptrs.size();
	float3 leftminextent;
	float3 leftextent = getAbsoluteExtent(left_prim_ptrs.data(), left.primitives_count, leftminextent);
	left.m_BoundingBox = Bounds3f(leftminextent, leftminextent + leftextent);

	right.primitives_count = right_prim_ptrs.size();
	float3 rightminextent;
	float3 rightextent = getAbsoluteExtent(right_prim_ptrs.data(), right.primitives_count, rightminextent);
	right.m_BoundingBox = Bounds3f(rightminextent, rightminextent + rightextent);
}

void BVHBuilder::makePartition(const Triangle** primitives_ptrs_buffer, size_t primitives_count, BVHNode& leftnode, BVHNode& rightnode)
{
	printf("partition input prim count:%zu \n", primitives_count);
	float lowestcost_partition_pt = 0;//best bin
	PARTITION_AXIS bestpartitionaxis{};

	int lowestcost = INT_MAX;

	float3 minextent = { FLT_MAX,FLT_MAX,FLT_MAX };
	float3 extent = getAbsoluteExtent(primitives_ptrs_buffer, primitives_count, minextent);
	Bounds3f parentbbox(minextent, minextent + extent);

	BVHNode left, right;

	//for x
	std::vector<float>bins;//world space
	bins.reserve(m_BinCount);
	float deltapartition = extent.x / m_BinCount;
	for (int i = 1; i < m_BinCount; i++)
	{
		bins.push_back(minextent.x + (i * deltapartition));
	}
	for (float bin : bins)
	{
		//printf("proc x bin %.3f\n", bin);
		binToShallowNodes(left, right, bin, PARTITION_AXIS::X_AXIS, primitives_ptrs_buffer, primitives_count);
		int cost = BVHNode::trav_cost + ((left.getSurfaceArea() / parentbbox.getSurfaceArea()) * left.primitives_count * left.rayint_cost) +
			((right.getSurfaceArea() / parentbbox.getSurfaceArea()) * right.primitives_count * right.rayint_cost);
		if (cost < lowestcost)
		{
			lowestcost = cost;
			bestpartitionaxis = PARTITION_AXIS::X_AXIS;
			lowestcost_partition_pt = bin;
		}
		left.Cleanup();
		right.Cleanup();
	}

	//for y
	bins.clear();
	deltapartition = extent.y / m_BinCount;
	for (int i = 1; i < m_BinCount; i++)
	{
		bins.push_back(minextent.y + (i * deltapartition));
	}
	for (float bin : bins)
	{
		//printf("proc y bin %.3f\n", bin);
		binToShallowNodes(left, right, bin, PARTITION_AXIS::Y_AXIS, primitives_ptrs_buffer, primitives_count);
		int cost = BVHNode::trav_cost + ((left.getSurfaceArea() / parentbbox.getSurfaceArea()) * left.primitives_count * left.rayint_cost) +
			((right.getSurfaceArea() / parentbbox.getSurfaceArea()) * right.primitives_count * right.rayint_cost);
		if (cost < lowestcost)
		{
			lowestcost = cost;
			bestpartitionaxis = PARTITION_AXIS::Y_AXIS;
			lowestcost_partition_pt = bin;
		}
		left.Cleanup();
		right.Cleanup();
	}

	//for z
	bins.clear();
	deltapartition = extent.z / m_BinCount;
	for (int i = 1; i < m_BinCount; i++)
	{
		bins.push_back(minextent.z + (i * deltapartition));
	}
	for (float bin : bins)
	{
		//printf("proc z bin %.3f\n", bin);
		binToShallowNodes(left, right, bin, PARTITION_AXIS::Z_AXIS, primitives_ptrs_buffer, primitives_count);
		int cost = BVHNode::trav_cost + ((left.getSurfaceArea() / parentbbox.getSurfaceArea()) * left.primitives_count * left.rayint_cost) +
			((right.getSurfaceArea() / parentbbox.getSurfaceArea()) * right.primitives_count * right.rayint_cost);
		if (cost < lowestcost)
		{
			lowestcost = cost;
			bestpartitionaxis = PARTITION_AXIS::Z_AXIS;
			lowestcost_partition_pt = bin;
		}
		left.Cleanup();
		right.Cleanup();
	}

	printf("made a partition, bin: %.3f, axis: %d, cost: %d\n", lowestcost_partition_pt, bestpartitionaxis, lowestcost);
	binToNodes(leftnode, rightnode, lowestcost_partition_pt,
		bestpartitionaxis, primitives_ptrs_buffer, primitives_count);
	printf("left node prim count:%d | right node prim count: %d\n", leftnode.primitives_count, rightnode.primitives_count);
}