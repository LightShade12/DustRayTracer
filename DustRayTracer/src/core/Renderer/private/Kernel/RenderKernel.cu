#include "hip/hip_runtime.h"
#include "RenderKernel.hpp"

#include "core/Renderer/private/Shapes/Scene.cuh"
#include "core/Renderer/private/Camera/Camera.hpp"
#include "core/Renderer/private/CudaMath/helper_math.hpp"//check if this requires definition activation

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#define __HIPCC__ // used to get surf2d indirect functions;not how it should be done
#include <surface_indirect_functions.h>

struct Ray {
	float3 origin;
	float3 direction;
};

struct Triangle {
	float3 vertex0, vertex1, vertex2;
};

__device__ bool rayTriangleIntersect(const Ray& ray, const Triangle& triangle, float& t) {
	const float EPSILON = 0.0000001;
	float3 edge1, edge2, h, s, q;
	float a, f, u, v;

	edge1.x = triangle.vertex1.x - triangle.vertex0.x;
	edge1.y = triangle.vertex1.y - triangle.vertex0.y;
	edge1.z = triangle.vertex1.z - triangle.vertex0.z;
	edge2.x = triangle.vertex2.x - triangle.vertex0.x;
	edge2.y = triangle.vertex2.y - triangle.vertex0.y;
	edge2.z = triangle.vertex2.z - triangle.vertex0.z;

	h = cross(ray.direction, edge2);
	a = dot(edge1, h);

	if (a > -EPSILON && a < EPSILON)
		return false; // This ray is parallel to this triangle.

	f = 1.0 / a;
	s.x = ray.origin.x - triangle.vertex0.x;
	s.y = ray.origin.y - triangle.vertex0.y;
	s.z = ray.origin.z - triangle.vertex0.z;
	u = f * dot(s, h);

	if (u < 0.0 || u > 1.0)
		return false;

	q = cross(s, edge1);
	v = f * dot(ray.direction, q);

	if (v < 0.0 || u + v > 1.0)
		return false;

	// At this stage we can compute t to find out where the intersection point is on the line.
	t = f * dot(edge2, q);
	if (t > EPSILON) // ray intersection
		return true;

	return false; // This means that there is a line intersection but not a ray intersection.
}

//Render Kernel
__global__ void kernel(hipSurfaceObject_t _surfobj, int max_x, int max_y, Camera* cam, const Sphere* sceneVector, size_t sceneVectorSize)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;

	float2 uv = { (float(i) / max_x) ,(float(j) / max_y) };

	//uv.x *= ((float)max_x / (float)max_y);
	//uv.x = uv.x * 2.f - ((float)max_x / (float)max_y);
	//uv.y = uv.y * 2.f - 1.f;
	uv = uv * 2 - 1;

	float3 rayOrigin = (cam)->m_Position;
	float3 rayDirection = cam->GetRayDir(uv, 30, max_x, max_y);
	//float radius = 0.5f;

	uchar4 color = { 0,0,0,255 };

	if (sceneVectorSize == 0)
	{
		float a = 0.5 * (1 + (normalize(rayDirection)).y);

		float3 col1 = { 0.5,0.7,1.0 };
		float3 col2 = { 1,1,1 };
		float3 fcol = (float(1 - a) * col2) + (a * col1);
		color.x = 255 * fcol.x;
		color.y = 255 * fcol.y;
		color.z = 255 * fcol.z;
		surf2Dwrite(color, _surfobj, i * 4, j);
		return;
	}

	const Sphere* closestSphere = nullptr;
	float hitDistance = FLT_MAX;//std::numeric_limits<float>::max();

	for (int i = 0; i < sceneVectorSize; i++)
	{
		const Sphere sphere = sceneVector[i];
		float3 origin = rayOrigin - sphere.Position;

		float a = dot(rayDirection, rayDirection);
		float b = 2.0f * dot(origin, rayDirection);
		float c = dot(origin, origin) - sphere.Radius * sphere.Radius;

		// Quadratic forumula discriminant:
		// b^2 - 4ac

		float discriminant = b * b - 4.0f * a * c;
		if (discriminant < 0.0f)
			continue;

		// Quadratic formula:
		// (-b +- sqrt(discriminant)) / 2a

		// float t0 = (-b + glm::sqrt(discriminant)) / (2.0f * a); // Second hit distance (currently unused)
		float closestT = (-b - sqrt(discriminant)) / (2.0f * a);
		if (closestT < hitDistance)
		{
			hitDistance = closestT;
			closestSphere = &sphere;
		}
	}

	if (closestSphere == nullptr)
	{
		float a = 0.5 * (1 + (normalize(rayDirection)).y);

		float3 col1 = { 0.5,0.7,1.0 };
		float3 col2 = { 1,1,1 };
		float3 fcol = (float(1 - a) * col2) + (a * col1);
		color.x = 255 * fcol.x;
		color.y = 255 * fcol.y;
		color.z = 255 * fcol.z;
		surf2Dwrite(color, _surfobj, i * 4, j);
		return;
	}

	float3 origin = rayOrigin - closestSphere->Position;//apply sphere translation
	float3 hitPoint = origin + rayDirection * hitDistance;
	float3 normal = normalize(hitPoint);
	
	float3 lightDir = normalize(make_float3(-1, -1, -1));
	float lightIntensity = max(dot(normal, -lightDir), 0.0f); // == cos(angle)

	float3 sphereColor = closestSphere->Albedo;
	sphereColor *= lightIntensity;

	color.x = 255 * sphereColor.x;
	color.y = 255 * sphereColor.y;
	color.z = 255 * sphereColor.z;

	surf2Dwrite(color, _surfobj, i * 4, j);
};

void InvokeRenderKernel(
	hipSurfaceObject_t surfaceobj, uint32_t width, uint32_t height,
	dim3 _blocks, dim3 _threads, Camera* cam, const Scene& scene)
{
	const Sphere* DeviceSceneVector = thrust::raw_pointer_cast(scene.m_Spheres.data());
	kernel << < _blocks, _threads >> > (surfaceobj, width, height, cam, DeviceSceneVector, scene.m_Spheres.size());
}