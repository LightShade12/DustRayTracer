#include "hip/hip_runtime.h"
#include "RenderKernel.cuh"

#include "core/Renderer/private/Kernel/Ray.cuh"
#include "core/Renderer/private/Kernel/HitPayload.cuh"
#include "core/Renderer/private/Shapes/Scene.cuh"
#include "core/Renderer/private/Camera/Camera.cuh"
#include "core/Renderer/private/CudaMath/helper_math.cuh"//check if this requires definition activation
#include "core/Renderer/private/CudaMath/Random.cuh"

#include "Shaders/RayGen.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#define __HIPCC__ // used to get surf2d indirect functions;not how it should be done
#include <surface_indirect_functions.h>

//Render Kernel
__global__ void kernel(hipSurfaceObject_t _surfobj, int max_x, int max_y, Camera* cam,
	const Triangle* sceneVector, size_t sceneVectorSize,
	const Material* materialvector, uint32_t frameidx, float3* accumulation_buffer)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= max_x) || (j >= max_y)) return;

	float3 fcolor = RayGen(i, j, max_x, max_y, cam, sceneVector,
		sceneVectorSize, materialvector, frameidx);

	accumulation_buffer[i + j * max_x] += fcolor;
	float3 accol = accumulation_buffer[i + j * max_x] / frameidx;
	uchar4 color = { unsigned char(255 * accol.x),unsigned char(255 * accol.y),unsigned char(255 * accol.z), 255 };

	surf2Dwrite(color, _surfobj, i * 4, j);
};

void InvokeRenderKernel(
	hipSurfaceObject_t surfaceobj, uint32_t width, uint32_t height,
	dim3 _blocks, dim3 _threads, Camera* cam, const Scene& scene, uint32_t frameidx, float3* accumulation_buffer)
{
	const Material* DeviceMaterialVector = thrust::raw_pointer_cast(scene.m_Material.data());;
	const Triangle* DeviceSceneVector = thrust::raw_pointer_cast(scene.m_Triangles.data());
	const Mesh* DeviceMeshBuffer = thrust::raw_pointer_cast(scene.m_Meshes.data());

	kernel << < _blocks, _threads >> >
		(surfaceobj, width, height, cam, DeviceSceneVector, scene.m_Triangles.size(),
			DeviceMaterialVector, frameidx, accumulation_buffer);
}