#include "hip/hip_runtime.h"
#include "RenderKernel.cuh"

#include "core/Renderer/private/Kernel/Ray.cuh"
#include "core/Renderer/private/Kernel/HitPayload.cuh"
#include "core/Renderer/private/Shapes/Scene.cuh"
#include "core/Renderer/private/Camera/Camera.cuh"
#include "core/Renderer/private/CudaMath/helper_math.cuh"//check if this requires definition activation

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#define __HIPCC__ // used to get surf2d indirect functions;not how it should be done
#include <surface_indirect_functions.h>

__device__ HitPayload ClosestHit(const Ray& ray, uint32_t obj_idx, float hit_distance, const Sphere* scene_vec) {
	const Sphere* sphere = &(scene_vec[obj_idx]);

	float3 origin = ray.origin - sphere->Position;//apply sphere translation

	HitPayload payload;
	payload.hit_distance = hit_distance;
	payload.world_position = origin + ray.direction * hit_distance;//hit position
	payload.world_normal = normalize(payload.world_position);
	payload.object_idx = obj_idx;

	payload.world_position += sphere->Position;

	return payload;
};

__device__ HitPayload Miss(const Ray& ray) {
	HitPayload payload;
	payload.hit_distance = -1;
	return payload;
};

__device__ HitPayload TraceRay(const Ray& ray, const Sphere* scene_vector, size_t scene_vector_size) {
	int closestObjectIdx = -1;
	float hitDistance = FLT_MAX;

	for (int i = 0; i < scene_vector_size; i++)
	{
		const Sphere* sphere = &scene_vector[i];
		float3 origin = ray.origin - sphere->Position;

		float a = dot(ray.direction, ray.direction);
		float b = 2.0f * dot(origin, ray.direction);
		float c = dot(origin, origin) - sphere->Radius * sphere->Radius;

		float discriminant = b * b - 4.0f * a * c;
		if (discriminant < 0.0f)
			continue;

		float closestT = (-b - sqrt(discriminant)) / (2.0f * a);
		if (closestT < hitDistance && closestT>0)
		{
			hitDistance = closestT;
			closestObjectIdx = i;
		}
	}

	if (closestObjectIdx < 0)
	{
		return Miss(ray);
	}

	return ClosestHit(ray, closestObjectIdx, hitDistance, scene_vector);
};

__device__ float3 RayGen(uint32_t x, uint32_t y, uint32_t max_x, uint32_t max_y,
	const Camera* cam, const Sphere* scene_vector, size_t scenevecsize) {
	float2 uv = { (float(x) / max_x) ,(float(y) / max_y) };

	//uv.x *= ((float)max_x / (float)max_y);
	//uv.x = uv.x * 2.f - ((float)max_x / (float)max_y);
	//uv.y = uv.y * 2.f - 1.f;
	uv = uv * 2 - 1;

	Ray ray;
	ray.origin = cam->m_Position;
	ray.direction = cam->GetRayDir(uv, 30, max_x, max_y);
	float3 color = { 0,0,0 };

	float multiplier = 1.f;
	int bounces = 2;
	for (int i = 0; i < bounces; i++)
	{
		HitPayload payload = TraceRay(ray, scene_vector, scenevecsize);
		//sky
		if (payload.hit_distance < 0)
		{
			float a = 0.5 * (1 + (normalize(ray.direction)).y);
			float3 col1 = { 0.5,0.7,1.0 };
			float3 col2 = { 1,1,1 };
			float3 fcol = (float(1 - a) * col2) + (a * col1);
			fcol = { 0,0,0 };
			color += fcol * multiplier;
			break;
		}

		float3 lightDir = normalize(make_float3(-1, -1, -1));
		float lightIntensity = max(dot(payload.world_normal, -lightDir), 0.0f); // == cos(angle)
		

		float3 spherecolor = scene_vector[payload.object_idx].Albedo;
		spherecolor *= lightIntensity;
		color += spherecolor * multiplier;
		
		multiplier *= 0.7f;

		ray.origin = payload.world_position + (payload.world_normal * 0.0001f);
		ray.direction = reflect(ray.direction, payload.world_normal);
		
		//color = { payload.world_normal.x, payload.world_normal.y, payload.world_normal.z };//debug normals
	}

	color = fminf(color, {1,1,1});
	return color;
};
//Render Kernel
__global__ void kernel(hipSurfaceObject_t _surfobj, int max_x, int max_y, Camera* cam, const Sphere* sceneVector, size_t sceneVectorSize)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;

	float3 fcolor = RayGen(i, j, max_x, max_y, cam, sceneVector, sceneVectorSize);
	uchar4 color = { unsigned char(255 * fcolor.x),unsigned char(255 * fcolor.y),unsigned char(255 * fcolor.z), 255 };

	surf2Dwrite(color, _surfobj, i * 4, j);
};

void InvokeRenderKernel(
	hipSurfaceObject_t surfaceobj, uint32_t width, uint32_t height,
	dim3 _blocks, dim3 _threads, Camera* cam, const Scene& scene)
{
	const Sphere* DeviceSceneVector = thrust::raw_pointer_cast(scene.m_Spheres.data());
	kernel << < _blocks, _threads >> > (surfaceobj, width, height, cam, DeviceSceneVector, scene.m_Spheres.size());
}