#include "hip/hip_runtime.h"
#include "RenderKernel.cuh"

#include "core/Renderer/private/Kernel/Ray.cuh"
#include "core/Renderer/private/Kernel/HitPayload.cuh"
#include "core/Renderer/private/Shapes/Scene.cuh"
#include "core/Renderer/private/Camera/Camera.cuh"
#include "core/Renderer/private/CudaMath/helper_math.cuh"//check if this requires definition activation
#include "core/Renderer/private/CudaMath/Random.cuh"

#include "Shaders/RayGen.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#define __HIPCC__ // used to get surf2d indirect functions;not how it should be done
#include <surface_indirect_functions.h>

//Render Kernel
__global__ void kernel(hipSurfaceObject_t _surfobj, int max_x, int max_y, Camera* cam, uint32_t frameidx, float3* accumulation_buffer, const SceneData scenedata)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= max_x) || (j >= max_y)) return;

	float3 fcolor = RayGen(i, j, max_x, max_y, cam, frameidx, scenedata);

	accumulation_buffer[i + j * max_x] += fcolor;
	float3 accol = accumulation_buffer[i + j * max_x] / frameidx;
	uchar4 color = { unsigned char(255 * accol.x),unsigned char(255 * accol.y),unsigned char(255 * accol.z), 255 };

	surf2Dwrite(color, _surfobj, i * 4, j);
};

void InvokeRenderKernel(
	hipSurfaceObject_t surfaceobj, uint32_t width, uint32_t height,
	dim3 _blocks, dim3 _threads, Camera* cam, const Scene& scene, uint32_t frameidx, float3* accumulation_buffer)
{
	SceneData scenedata;
	scenedata.DeviceTextureBufferPtr = thrust::raw_pointer_cast(scene.m_Textures.data());
	scenedata.DeviceMeshBufferPtr = thrust::raw_pointer_cast(scene.m_Meshes.data());
	scenedata.DeviceMaterialBufferPtr = thrust::raw_pointer_cast(scene.m_Material.data());
	scenedata.DeviceMeshBufferSize = scene.m_Meshes.size();

	kernel << < _blocks, _threads >> > (surfaceobj, width, height, cam, frameidx, accumulation_buffer, scenedata);
}