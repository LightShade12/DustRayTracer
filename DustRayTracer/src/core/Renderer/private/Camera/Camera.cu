#include "hip/hip_runtime.h"
#include "Camera.hpp"

__device__ void Camera::Translate(float3 movedir)
{
	m_Position += movedir.z * m_Forward_dir;
	m_Position += movedir.y * m_Up_dir;
	m_Position += movedir.x * m_Right_dir;
}

__device__ float deg2rad(float degree)
{
	float const PI = 3.14159265359;
	return (degree * (PI / 180));
}


/*
camera(float vfov, glm::vec3 lookfrom, glm::vec3 lookdir, glm::vec3 vup, float aperture, float focus_dist) {
	auto theta = degrees_to_radians(vfov);
	auto h = tan(theta / 2);
	float viewport_height = 2.0 * h;
	float viewport_width = viewport_height;

	auto focal_length = 1.0;

	w = glm::normalize(lookdir);
	u = glm::normalize(cross(vup, w));
	v = cross(w, u);

	origin = lookfrom;
	horizontal = focus_dist * viewport_width * u;
	vertical = focus_dist * viewport_height * v;
	lower_left_corner = origin - horizontal / 2.0f - vertical / 2.0f - focus_dist * w;

	lens_radius = aperture / 2;
}

ray get_ray(float s, float t) const {
	glm::vec3 rd = lens_radius * random_in_unit_disk();
	glm::vec3 offset = u * rd.x + v * rd.y;

	return ray(origin + offset, lower_left_corner + s * horizontal + t * vertical - origin - offset);
}
public:
	glm::vec3 w;//front
	glm::vec3 u;//right
	glm::vec3 v;//up
	glm::vec3 origin;
	glm::vec3 lower_left_corner;
	glm::vec3 horizontal;
	glm::vec3 vertical;
*/


__device__ float3 Camera::GetRayDir(float2 _uv, float vfovdeg, float width, float height)
{
	float theta = deg2rad(vfovdeg);
	auto h = tan(theta / 2);
	float viewport_height = 2.0 * h;
	float viewport_width = viewport_height * (width / height);//aspect ratio
	//float viewport_width = viewport_height;

	float3 w = normalize(m_Forward_dir);//front
	float3 u = normalize(cross(w, make_float3(0, 1, 0)));//right
	float3 v = cross(u, w);//up

	float3 horizontal = viewport_width * u;
	float3 vertical = viewport_height * v;

	float3 lower_left_corner = m_Position - horizontal / 2.0f - vertical / 2.0f + w;

	//return float3(lower_left_corner + _uv.x * horizontal + _uv.y * vertical - m_Position);
	return float3((m_Position + w) + _uv.x * horizontal + _uv.y * vertical - m_Position);
}